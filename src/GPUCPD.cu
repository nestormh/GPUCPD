/*
    Copyright 2013 Néstor Morales Hernández <email>

    Licensed under the Apache License, Version 2.0 (the "License");
    you may not use this file except in compliance with the License.
    You may obtain a copy of the License at

        http://www.apache.org/licenses/LICENSE-2.0

    Unless required by applicable law or agreed to in writing, software
    distributed under the License is distributed on an "AS IS" BASIS,
    WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
    See the License for the specific language governing permissions and
    limitations under the License.
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <string>
#include <stdexcept>

#include <hipblas.h>

#define PI 3.14159265358979

using namespace std;

extern "C"
void checkCudaError(const hipError_t& hipError_t, const int & line)
{
    if (hipError_t != hipSuccess) {
        char errorStr[1024];
        sprintf(errorStr, "Error %s, at line %s:%d", hipGetErrorString(hipError_t), __FILE__, line);
        throw (std::runtime_error(errorStr));
    }
}

extern "C"
void checkCublasReturn(const hipblasStatus_t& retCublas, const int & line)
{
    if (retCublas != HIPBLAS_STATUS_SUCCESS) {
        char statStr[1024];
        sprintf(statStr, "Error in cublas calculation with code %d, at line %s:%d", retCublas, __FILE__, line);
        throw std::runtime_error(statStr);
    }
}

__global__
void sumRows(const float * A, const uint32_t & M, const uint32_t & N, float * S) {
    const unsigned int idxM = (blockIdx.x * blockDim.x + threadIdx.x);
    const unsigned int idxN = (blockIdx.y * blockDim.y + threadIdx.y);
    const unsigned int idx = idxN * M + idxM;
    
    if ((idxM >= M) || (idxN >= N))
        return;
}

extern "C"
void launchSumRows(const float * A, const uint32_t & M, const uint32_t & N, float * S) {
    
}

__global__
void getG(float * d_G, const float * d_Y, const unsigned int M, const unsigned int D, 
          const float gamma) {
            
    const unsigned int idxI = (blockIdx.x * blockDim.x + threadIdx.x);
    const unsigned int idxJ = (blockIdx.y * blockDim.y + threadIdx.y);
    
    if ((idxI > idxJ) || (idxI >= M) || (idxJ >= M))
        return;
    
    const unsigned int idxG = idxI * M + idxJ;
    const unsigned int idxGinv = idxJ * M + idxI;

    float value = 0.0;
    float term;
    for (int i = 0; i < D; i++) {
        term = d_Y[i * M + idxI] - d_Y[i * M + idxJ];
        value += term * term;
    }
    value = exp(value * gamma);

    d_G[idxG] = value;
    d_G[idxGinv] = value;
}

__global__
void getNumerator(const float * d_X, const float * d_YGW, float * d_numerator, const unsigned int M, const unsigned int N, 
                  const unsigned int D, const float gamma) {
    
    const unsigned int idxM = (blockIdx.x * blockDim.x + threadIdx.x);
    const unsigned int idxN = (blockIdx.y * blockDim.y + threadIdx.y);
    const unsigned int idx = idxN * M + idxM;
    
    if ((idxM >= M) || (idxN >= N))
        return;
    
    float value = 0.0;
    float term;
    for (int i = 0; i < D; i++) {
        term = d_X[i * N + idxN] - d_YGW[i * M + idxM];
        value += term * term;
    }
    value *= gamma;
    
    d_numerator[idx] = exp(value);
}

__global__
void getP(const float * d_numerator, const float * d_denominator, float * d_P, const unsigned int M, const unsigned int N, const float omegaTmp) {
    
    const unsigned int idxM = (blockIdx.x * blockDim.x + threadIdx.x);
    const unsigned int idxN = (blockIdx.y * blockDim.y + threadIdx.y);
    const unsigned int idx = idxN * M + idxM;
    
    if ((idxM >= M) || (idxN >= N))
        return;

    if ((idxM == 1) && (idxN == 1))  {
        printf("P[%d,%d = %d] = %f / %f = %f\n", idxM, idxN, idx, d_numerator[idx], d_denominator[idxN], d_numerator[idx] / d_denominator[idxN]);
    }
    d_P[idx] = d_numerator[idx] / (d_denominator[idxN] + omegaTmp);
}
    
extern "C"
void launchGetG(float * d_G, const float * d_Y, const int & M, const int & D, 
                const float & beta, const hipDeviceProp_t & deviceProp) {
    
    const float gamma = -1 / (2 * beta * beta);
    
    int blockDimension = sqrt(deviceProp.maxThreadsPerBlock);
    int gridDimension = M / blockDimension + 1;
    
    const dim3 blockSize(blockDimension, blockDimension, 1);
    const dim3 gridSize(gridDimension, gridDimension, 1);
    
    struct timespec start, finish;
    double elapsed;
    clock_gettime(CLOCK_MONOTONIC, &start);
    
    getG <<<gridSize, blockSize>>> (d_G, d_Y, M, D, gamma);
    hipDeviceSynchronize(); hipGetLastError();
    
    clock_gettime(CLOCK_MONOTONIC, &finish);
    elapsed = (finish.tv_sec - start.tv_sec);
    elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;
    
    std::cout << "Elapsed time for getting G matrix = " << elapsed << endl;
    
}


extern "C"
void launchGetP(const float * d_X, const float * d_tmpMxD, const float * d_ones1xM, const float * d_ones1xN, float * d_tmpMxN, float * d_tmp1xN, float * d_P,
                const int & M, const int & N, const int & D, const float & omega, const float & sigma2,
                const hipDeviceProp_t & deviceProp, const hipblasHandle_t & cublasHandle) {
             
    const float gamma = -1 / (2 * sigma2);
    
    cout << "sigma2 = " << sigma2 << endl;
    cout << "gamma = " << gamma << endl;
    
    int blockDimension = sqrt(deviceProp.maxThreadsPerBlock);
    int gridDimensionX = N / blockDimension + 1;
    int gridDimensionY = M / blockDimension + 1;
    
    const dim3 blockSize(blockDimension, blockDimension, 1);
    const dim3 gridSize(gridDimensionX, gridDimensionY, 1);
    
    struct timespec start, finish;
    double elapsed;
    clock_gettime(CLOCK_MONOTONIC, &start);
    
    // First, we obtain the matrix for the numerator
    float * d_numerator = d_tmpMxN;
    const float * d_YGW = d_tmpMxD;
    getNumerator <<<gridSize, blockSize>>> (d_X, d_YGW, d_numerator, M, N, D, gamma);
    hipDeviceSynchronize(); hipGetLastError();

    // Now, we obtain the matrix for the denominator
    float * d_denominator = d_tmp1xN;
    {
        const float alpha = 1.0;
        const float beta = 0.0;
        checkCublasReturn(hipblasSgemv(cublasHandle, HIPBLAS_OP_T,
                                         M, N, &alpha, d_numerator, M, d_ones1xM, 1, &beta, d_denominator, 1), __LINE__);
        //FIXME Sum this in the GetP function!!!
//         const float term2denom = (omega / (1 - omega)) * ((pow(2 * PI * sigma2, D * 0.5) * M) / N);
//         checkCublasReturn(cublasSaxpy_v2(cublasHandle, N, &term2denom, d_ones1xN, 1, d_denominator, 1), __LINE__);
    }
    
    float omegaTmp = (omega * pow((2 * PI * sigma2), (0.5 * D)) * M) / ((1 - omega) * N);
    
    // Finally, we divide the numerator by the denominator
    {
        const dim3 blockSize(1, deviceProp.maxThreadsPerBlock, 1);

        int gridDimensionX = N;
        int gridDimensionY = M / blockSize.y + 1;
        
        const dim3 gridSize(gridDimensionX, gridDimensionY, 1);
        
        cout << "omegaTmp = " << omegaTmp << endl;
        
        getP <<<gridSize, blockSize>>> (d_numerator, d_denominator, d_P, M, N, omegaTmp);
        hipDeviceSynchronize(); hipGetLastError();
    }
    
    clock_gettime(CLOCK_MONOTONIC, &finish);
    elapsed = (finish.tv_sec - start.tv_sec);
    elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;
    
    std::cout << "Elapsed time for getting P matrix = " << elapsed << endl;
}